#include "hip/hip_runtime.h"


// System includes
#include <stdio.h>
#include <stdint.h>
#include <assert.h>
#include "utils.h"
#include <algorithm>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <sm_30_intrinsics.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define BLOCK_SIZE 1024
void Check_CUDA_Error(const char* message) {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "ERROR: %s: %s\n", message, hipGetErrorString(error));
        exit(-1);
    }
}
__global__ void inner_prod(const double* x, const double* y, double* sum, long N)
{
    __shared__ double smem_x[BLOCK_SIZE];
    __shared__ double smem_y[BLOCK_SIZE];
    __shared__ double smem_sum[BLOCK_SIZE];
    int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

    if (idx < N) {
        smem_x[threadIdx.x] = x[idx];
        smem_y[threadIdx.x] = y[idx];
    }
    else {
        smem_x[threadIdx.x] = 0;
        smem_y[threadIdx.x] = 0;
    }
    __syncthreads();
    if (threadIdx.x < 512) smem_sum[blockIdx.x] += smem_y[threadIdx.x + 512] * smem_x[threadIdx.x + 512];
    __syncthreads();
    if (threadIdx.x < 256) smem_sum[blockIdx.x] += smem_y[threadIdx.x + 256] * smem_x[threadIdx.x + 256];
    __syncthreads();
    if (threadIdx.x < 128) smem_sum[blockIdx.x] += smem_y[threadIdx.x + 128] * smem_x[threadIdx.x + 128];
    __syncthreads();
    if (threadIdx.x < 64) smem_sum[blockIdx.x] += smem_y[threadIdx.x + 64] * smem_x[threadIdx.x + 64];
    __syncthreads();
    if (threadIdx.x < 32) {
        smem_sum[blockIdx.x] += smem_y[threadIdx.x + 32] * smem_x[threadIdx.x + 32];
        __syncwarp();
        smem_sum[blockIdx.x] += smem_y[threadIdx.x + 16] * smem_x[threadIdx.x + 16];
        __syncwarp();
        smem_sum[blockIdx.x] += smem_y[threadIdx.x + 8] * smem_x[threadIdx.x + 8];
        __syncwarp();
        smem_sum[blockIdx.x] += smem_y[threadIdx.x + 4] * smem_x[threadIdx.x + 4];
        __syncwarp();
        smem_sum[blockIdx.x] += smem_y[threadIdx.x + 2] * smem_x[threadIdx.x + 2];
        __syncwarp();
        if (threadIdx.x == 0) smem_sum[blockIdx.x] += smem_sum[0]+smem_sum[1];
   

        //printf("hello from block %d and thread %d / %d \n", blockIdx.x, threadIdx.x, blockDim.x);
        //printf("inner product from GPU = %f \n", sum[blockIdx.x]);
    }
 
    *sum += smem_sum[blockIdx.x];
}
__global__ void reduction_kernel2(double* sum, const double* a, long N) {
    __shared__ double smem[BLOCK_SIZE];
    int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

    if (idx < N) smem[threadIdx.x] = a[idx];
    else smem[threadIdx.x] = 0;

    __syncthreads();
    if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
    __syncthreads();
    if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
    __syncthreads();
    if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
    __syncthreads();
    if (threadIdx.x < 64) smem[threadIdx.x] += smem[threadIdx.x + 64];
    __syncthreads();
    if (threadIdx.x < 32) {
        smem[threadIdx.x] += smem[threadIdx.x + 32];
        __syncwarp();
        smem[threadIdx.x] += smem[threadIdx.x + 16];
        __syncwarp();
        smem[threadIdx.x] += smem[threadIdx.x + 8];
        __syncwarp();
        smem[threadIdx.x] += smem[threadIdx.x + 4];
        __syncwarp();
        smem[threadIdx.x] += smem[threadIdx.x + 2];
        __syncwarp();
        if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
    }
}
double inner_prod_cpu(double *x, double *y, long N) {
    double sum = 0.0;
#pragma omp parallel for schedule(static) reduction(+:sum)
    for (int i = 0; i < N; i++) {
        sum += x[i] * y[i];
    }
    return sum;
}

int main(int argc, char **argv)
{
    // filling arrays for inner product
    long const N = 100;
    double* x;
    double* y;
    hipHostMalloc((void**)&x, N * sizeof(double));
    hipHostMalloc((void**)&y, N * sizeof(double));
    
#pragma omp parallel for schedule(static)
    for (int i = 0; i < N; i++) {
        x[i] = 1.0;
        y[i] = 1.0;
    }

    Timer tt;
    // CPU version, for reference
    tt.tic();
    double sum_ref = inner_prod_cpu(x, y, N);
    printf("CPU: t = %f seconds ; sum = %f \n", tt.toc(), sum_ref);

    // allocating memory for GPU
    double sum;
    double *x_d, *y_d, *sum_d;
    hipMalloc(&x_d, N * sizeof(double));
    hipMalloc(&y_d, N * sizeof(double));
    long N_work = 1;
    for (long i = (N + BLOCK_SIZE - 1) / (BLOCK_SIZE); i > 1; i = (i + BLOCK_SIZE - 1) / (BLOCK_SIZE)) N_work += i;
    hipMalloc(&sum_d, N_work * sizeof(double)); // extra memory buffer for reduction across thread-blocks

    // copying vectors to GPU
    hipMemcpyAsync(x_d, x, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(y_d, y, N * sizeof(double), hipMemcpyHostToDevice);
    
    hipDeviceSynchronize();
    long Nb = (N + BLOCK_SIZE - 1) / (BLOCK_SIZE);
    // GPU version
    tt.tic();

    // max thread number per block is BS = 1024 
    // << # of blocks, # of threads per block >>
    // # blocks =( N + BS -1 )/ BS; # threads/block = BS
    // so that # threads ~ N (if N > BS)
    inner_prod<<<Nb, BLOCK_SIZE>>>(x_d, y_d, sum_d, N);
    
    hipMemcpyAsync(&sum, sum_d, 1 * sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printf("GPU: t = %f seconds ; sum = %f \n", tt.toc(), sum);
    hipFree(x_d);
    hipFree(y_d);
    hipFree(sum_d);
    hipHostFree(x);
    hipHostFree(y);
    return 0;
}
